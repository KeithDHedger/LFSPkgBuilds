#include "hip/hip_runtime.h"
#include "cuMatrix.h"

__global__ void make_BlackWhite(int *image, int N){
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	image[y*N + x] = image[y*N + x] > 128 ? 255 : 0;
}

void convertToArray(int **matrix, int *array, int N){
	for(unsigned int i=0; i< N; i++)
		for(unsigned int j=0; j< N; j++)
			array[i*N+ j] = matrix[i][j];
}

// Wrapper function for kernel launch (not the complete function, just an example).
template <class T>
void
reduce(int size, int threads, int blocks,
       int whichKernel, T *d_idata, T *d_odata)
{
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);

	// when there is only one warp per block, we need to allocate two warps
	// worth of shared memory so that we don't index shared memory out of bounds
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

	// choose which of the optimized versions of reduction to launch
	switch (whichKernel)
	{
		case 0:
			reduce0<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
			break;

		case 1:
			reduce1<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
			break;
	}
}
